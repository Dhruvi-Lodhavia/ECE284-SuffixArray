#include "hip/hip_runtime.h"
#include "seedTable.cuh"
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <math.h>
// #include <lock.h>

/**
 * Prints information for each available GPU device on stdout
 */
void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    // For each GPU device found, print the information (memory, bandwidth etc.)
    // about the device
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

/**
 * Allocates arrays on the GPU device for (i) storing the compressed sequence
 * (ii) kmer offsets of the seed table (iii) kmer positions of the seed table
 * Size of the arrays depends on the input sequence length and kmer size
 */
void GpuSeedTable::DeviceArrays::allocateDeviceArrays (uint32_t* compressedSeq, uint32_t seqLen, uint32_t kmerSize) {
    hipError_t err;

    d_seqLen = seqLen;
    uint32_t compressedSeqLen = (seqLen+15)/16;
    uint32_t maxKmers = (uint32_t) pow(4,kmerSize)+1;
    

    // Only (1)allocate and (2)transfer the 2-bit compressed sequence to GPU.
    // This reduces the memory transfer and storage overheads
    // 1. Allocate memory
    err = hipMalloc(&d_compressedSeq, compressedSeqLen*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // 2. Transfer compressed sequence
    err = hipMemcpy(d_compressedSeq, compressedSeq, compressedSeqLen*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer offset array
    err = hipMalloc(&d_array2, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer offset array
    err = hipMalloc(&d_intermediate_array, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer offset array
    err = hipMalloc(&d_intermediate_array2, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer position array
    // Each element is size_t (64-bit) because an intermediate step uses the
    // first 32-bits for kmer value and the last 32-bits for kmer positions
    err = hipMalloc(&d_array1, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    // hipDeviceSynchronize();

    err = hipMalloc(&d_array3, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_suffix_array, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_done, (1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    hipDeviceSynchronize();
}

/**
 * Free allocated GPU device memory for different arrays
 */
void GpuSeedTable::DeviceArrays::deallocateDeviceArrays () {
    hipFree(d_compressedSeq);
    hipFree(d_array2);
    hipFree(d_intermediate_array);
    hipFree(d_intermediate_array2);
    hipFree(d_array1);
    hipFree(d_array3);
    hipFree(d_suffix_array);
    hipFree(d_done);
}

/**
 * Finds kmers for the compressed sequence creates an array with elements
 * containing the 64-bit concatenated value consisting of the kmer value in the
 * first 32 bits and the kmer position in the last 32 bits. The values are
 * stored in the arrary kmerPos, with i-th element corresponding to the i-th
 * kmer in the sequence
 *
 * ASSIGNMENT 2 TASK: parallelize this function
 */
__global__ void kmerPosConcat(
    uint32_t* d_compressedSeq,
    uint32_t d_seqLen,
    uint32_t kmerSize,
    size_t* d_array1) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    // HINT: Values below could be useful for parallelizing the code
    int bs = blockDim.x;
    int gs = gridDim.x;

    int i = bs*bx+tx;
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;

    // Helps mask the non kmer bits from compressed sequence. E.g. for k=2,
    // mask=0x1111 and for k=3, mask=0x111111
    uint32_t mask = (1 << 2*k)-1;
    size_t kmer = 0;
    
    // HINT: the if statement below ensures only the first thread of the first
    // block does all the computation. This statement might have to be removed
    // during parallelization
    // if ((bx == 0) && (tx == 0)) {
    // for (uint32_t i = 0; i <= N-k; i++) {
    
    while(i<=N-k){
        uint32_t index = i/16;
        uint32_t shift1 = 2*(i%16);
        if (shift1 > 0) {
            uint32_t shift2 = 32-shift1;
            kmer = ((d_compressedSeq[index] >> shift1) | (d_compressedSeq[index+1] << shift2)) & mask;
        } else {
            kmer = d_compressedSeq[index] & mask;
        }

        // Concatenate kmer value (first 32-bits) with its position (last
        // 32-bits)
        size_t arrayConcat;
        
        arrayConcat = (kmer << 32) + i;
        
        d_array1[i] = arrayConcat;
        i+=bs*gs;
    }
    
}

/**
 * Generates the kmerOffset array using the sorted kmerPos array consisting of
 * the kmer and positions. Requires iterating through the kmerPos array and
 * finding indexes where the kmer values change, depending on which the
 * kmerOffset values are determined.
 *
 * ASSIGNMENT 2 TASK: parallelize this function
 */
__global__ void kmerOffsetFill(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    size_t* d_array1) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // HINT: Values below could be useful for parallelizing the code
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int ty = threadIdx.y;
    // int by = blockIdx.y;
    // HINT: Values below could be useful for parallelizing the code
    // int bsy = blockDim.y;
    // Lock myLock;
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;
   
    size_t mask = ((size_t) 1 << 32)-1;
    uint32_t kmer = 0;
    uint32_t lastKmer = 0;
    // uint32_t j = 0;
    // int i = bs*bx+tx;
  
    // HINT: the if statement below ensures only the first thread of the first
    // block does all the computation. This statement might have to be removed
    // during parallelization
    
    for (uint32_t i = (bx * bs + tx); i < N-k; i+=bs*gs){
        lastKmer = (d_array1[i] >> 32) & mask;
        kmer = (d_array1[i+1] >> 32) & mask;
        
        if(kmer == lastKmer){
            d_array2[i+1] = 0;
        }
        else{
            d_array2[i+1] = i+1;
        }   
    }   
}

__global__ void prefixsum(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    // size_t* d_intermediate_array,
    // size_t* d_kmerPos,
    size_t* d_array3,
    uint32_t range) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // HINT: Values below could be useful for parallelizing the code
    int bs = blockDim.x;
    int gs = gridDim.x;
    

    for(uint32_t index = bx; index < ((range+bs-1)/bs); index+=gs){ //loop1
        
        // __shared__ size_t array_shared[8]; //bs size
        __shared__ size_t array_shared[2048]; //bs size

        uint32_t startAddress = index*(bs);
        if((startAddress+tx) < range){
            array_shared[tx] = d_array2[startAddress + tx];
        }
        else{
            array_shared[tx] = 0;
        }
        __syncthreads();
        int n = bs;
        // int m = tx;
        // int offset = 1;
        // uint32_t mappingScore = 0;

        
        for (int offset=1; offset<n; offset*=2) {
            int val = (tx + 1) * offset * 2 - 1;
            if (val< n) {
                // kmerOffset_shared[val] += kmerOffset_shared[val - offset];
                array_shared[val] = max(array_shared[val - offset], array_shared[val]);
            }
            __syncthreads();
        }

        
        for (int offset=n/2; offset>0; offset>>= 1) {
            __syncthreads();
            int val = (tx + 1) * offset * 2 - 1;
            if (val < n) {
                // kmerOffset_shared[val+offset] += kmerOffset_shared[val];
                array_shared[val+offset] = max(array_shared[val],array_shared[val+offset]);
            }
        }
        __syncthreads();
        if((startAddress+ tx) < d_seqLen){
            d_array2[startAddress + tx] = array_shared[tx];
        }
        d_array3[index] = array_shared[n-1];
    }
}

__global__ void reductionStep(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    // size_t* d_intermediate_array,
    // size_t* d_kmerPos,
    size_t* d_array3,
    uint32_t range) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // HINT: Values below could be useful for parallelizing the code
    int bs = blockDim.x;
    int gs = gridDim.x;

    for(uint32_t index = bx; index< (range/bs)+1; index+=gs){ //loop3
        uint32_t startAddress = index*(bs);
        if(((startAddress+tx) < range) && (index!=0)){
            d_array2[startAddress + tx] = max(d_array3[index-1],d_array2[startAddress + tx]);
        }
        else{
            d_array2[startAddress + tx] += 0;
        }        
    }
}

    /**
    * Masks the first 32 bits of the elements in the kmerPos array
    *
    * ASSIGNMENT 2 TASK: parallelize this function
    */
    __global__ void kmerPosMask(
        uint32_t d_seqLen,
        uint32_t kmerSize,
        size_t* d_array1) {

        
        int tx = threadIdx.x;
        int bx = blockIdx.x;

        

        // HINT: Values below could be useful for parallelizing the code
        int bs = blockDim.x;
        int gs = gridDim.x;

        int i = bs*bx+tx;

        uint32_t N = d_seqLen;
        uint32_t k = kmerSize;

        size_t mask = ((size_t) 1 << 32)-1;
        // size_t kPosConcat = (kmer << 32) + i;
        
        while(i<=N-k){
            // // (d_kmerPos[i] >> 32) & mask;
            // size_t kmerPosConcat = ((d_kmerPos[i] & mask)<< 32)
            // d_kmerPos[i] = kmerPosConcat + d_kmerOffset[i];
            d_array1[i] = (d_array1[i] & mask);
            i+=bs*gs;
            }
}

__global__ void reordering(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    // size_t* d_intermediate_array,
    // size_t* d_kmerPos,
    size_t* d_array1,
    size_t* d_array3) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int i = bs*bx+tx;
    //kmerpos = SA
    //kmer offset = B
    //kmerpos = B'
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;
    
    for (uint32_t i = (bx * bs + tx); i <= N-k; i+=bs*gs){
        uint32_t new_index = d_array1[i];
        d_array3[new_index] = d_array2[i];
    } 
}

__global__ void shifting(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array1,
    size_t* d_array3,
    uint32_t shift_val,
    size_t* d_suffix_array) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int i = bs*bx+tx;
    //kmerpos = SA
    //kmer offset = B
    //kmerpos = B'
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;

    // for(uint32_t index = bx; index < ((N+bs-1)/bs); index+=gs){ //loop1
        
    //     __shared__ size_t temp_memory[8]; //bs size
    //     // __shared__ size_t temp_memory[2048]; //bs size

    //     uint32_t startAddress = index*(bs);
    //     if((startAddress+tx) < N-shift_val){
    //         temp_memory[tx] = d_array3[startAddress + tx +shift_val];
    //     }
    //     else{
    //         temp_memory[tx] = 0;
    //     }
    //     __syncthreads();

    //     d_array1[startAddress +tx] = temp_memory[tx];

    // }


    //need to fill with 0s initially or atleast the shifted positions
    for (uint32_t i = (bx * bs + tx); i <= N-k; i+=bs*gs){
        d_suffix_array[i] = i;
        if(i<=N-1-shift_val){
            d_array1[i] = d_array3[i+shift_val];
        }
        else{
            d_array1[i] = 0;
        }
    } 
}

__global__ void merging(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array1,
    size_t* d_array3) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int i = bs*bx+tx;
    //kmerpos = SA
    //kmer offset = B
    //kmerpos = B'
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;
    //need to fill with 0s initially or atleast the shifted positions
    for (uint32_t i = (bx * bs + tx); i <= N-k; i+=bs*gs){
        d_array1[i] += d_array3[i]<<32;
    } 
}

__global__ void kmerOffsetFill2(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    size_t* d_array1) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // HINT: Values below could be useful for parallelizing the code
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int ty = threadIdx.y;
    // int by = blockIdx.y;
    // HINT: Values below could be useful for parallelizing the code
    // int bsy = blockDim.y;
    // Lock myLock;
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;
   
    size_t kmer = 0;
    size_t lastKmer = 0;
    // uint32_t j = 0;
    // int i = bs*bx+tx;
  
    // HINT: the if statement below ensures only the first thread of the first
    // block does all the computation. This statement might have to be removed
    // during parallelization
    
    for (uint32_t i = (bx * bs + tx); i < N-k; i+=bs*gs){
        lastKmer = d_array1[i];
        kmer = d_array1[i+1];
        
        if(kmer == lastKmer){
            d_array2[i+1] = 0;
        }
        else{
            d_array2[i+1] = i+1;
        }   
    }   
}

__global__ void singleton(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    size_t* d_array2,
    size_t* d_done){
    // uint32_t &done) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    // int i = bs*bx+tx;
    //kmerpos = SA
    //kmer offset = B
    //kmerpos = B'
    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;
    //need to fill with 0s initially or atleast the shifted positions

    uint32_t kmer = 0;
    uint32_t lastKmer = 0;
    size_t mask = ((size_t) 1 << 32)-1;
    if((bx==0) && (tx==0))
    {
        d_done[0] = 1;
    }
    for (uint32_t i = (bx * bs + tx); i < N-k; i+=bs*gs){
        lastKmer = d_array2[i];
        kmer = d_array2[i+1];
        if(kmer == lastKmer){
            d_done[0] = 0;
            return;
        }
        
    }
    // if((bx==0) && (tx==0))
    // {
    //     d_done[0] = 1;
    // }
}



/**
 * Constructs seed table, consisting of kmerOffset and kmerPos arrrays
 * on the GPU.
*/
void GpuSeedTable::seedTableOnGpu (
    uint32_t* compressedSeq,
    uint32_t seqLen,
    uint32_t kmerSize,
    size_t* array2,
    size_t* intermediate_array,
    size_t* intermediate_array2,
    size_t* array1,
    size_t* array3,
    size_t* suffix_array,
    size_t* done) {

    // // ASSIGNMENT 2 TASK: make sure to appropriately set the values below
    int numBlocks =  1024; // i.e. number of thread blocks on the GPU
    int blockSize = 1024; // i.e. number of GPU threads per thread block

    // int numBlocks =  2; // i.e. number of thread blocks on the GPU
    // int blockSize = 10; // i.e. number of GPU threads per thread block

    kmerPosConcat<<<numBlocks, blockSize>>>(compressedSeq, seqLen, kmerSize, suffix_array);

    // Parallel sort the kmerPos array on the GPU device using the thrust
    // library (https://thrust.github.io/)
    // thrust::device_ptr<size_t> array1Ptr(array1);
    thrust::device_ptr<size_t> interPtr(suffix_array);
    thrust::sort(interPtr, interPtr+seqLen-kmerSize+1);
    // thrust::device_ptr<size_t> array1Ptr(array1);
    // thrust::device_ptr<size_t> interPtr2(suffix_array);
    uint32_t numKmers = pow(4, kmerSize);
    uint32_t range = seqLen;
    // printf("range = %u",range);
    // printf("range2 = %u",num);
    // printf("range3 = %u",((num+blockSize-1)/blockSize));
    
    kmerOffsetFill<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,suffix_array);
    prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,array3,range);
    uint32_t num = ((range+blockSize-1)/blockSize);
    prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array3,intermediate_array,num);
    uint32_t num2 = ((num+blockSize-1)/blockSize);
    prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, intermediate_array,intermediate_array2,num2);
    reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, intermediate_array,intermediate_array2,((range/blockSize)/blockSize));
    reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array3,intermediate_array,(range/blockSize));
    reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,array3,range);

    kmerPosMask<<<numBlocks, blockSize>>>(seqLen, kmerSize, suffix_array);
    // uint32_t done= 0;
    // hipDeviceSynchronize();
    size_t* done2 = new size_t[1];
    size_t* SA_final = new size_t[seqLen-kmerSize+1];
    size_t* array2_final = new size_t[seqLen-kmerSize+1];
    size_t* array1_final = new size_t[seqLen-kmerSize+1];
    // uint32_t iter = 1;
    uint32_t shift_val = 1;
    uint32_t iteration = 0;
    do{ 
        // iteration+=1;
        reordering<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,suffix_array,array3);
        shifting<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers,array1,array3,shift_val,suffix_array);
        shift_val = shift_val<<1;
        merging<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers,array1,array3);

        thrust::device_ptr<size_t> array1Ptr(array1);
        thrust::device_ptr<size_t> suffixPtr(suffix_array);
        thrust::sort_by_key(array1Ptr, array1Ptr+seqLen-kmerSize+1,suffixPtr);

        kmerOffsetFill2<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,array1);
        hipMemcpy(array2_final, array2, (seqLen-kmerSize+1)*sizeof(size_t), hipMemcpyDeviceToHost);

        hipMemcpy(array1_final, array1, (seqLen-kmerSize+1)*sizeof(size_t), hipMemcpyDeviceToHost);

        // for (uint32_t i = 0; i <= seqLen-kmerSize; i++) {
        //     printf("array1[%u]=%lu\n", i, array1_final[i]);
        // }
        
        // for (uint32_t i = 0; i <= seqLen-kmerSize; i++) {
        //     printf("array2[%u]=%lu\n", i, array2_final[i]);
        // }
        prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,array3,range);

        prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array3,intermediate_array,num);
        prefixsum<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, intermediate_array,intermediate_array2,num2);
        reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, intermediate_array,intermediate_array2,((range/blockSize)/blockSize));
        reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array3,intermediate_array,(range/blockSize));
        reductionStep<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, array2,array3,range);
        
        
        singleton<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers,array2,done);
        
        
        

        // hipMemcpy(SA_final, suffix_array, (seqLen-kmerSize+1)*sizeof(size_t), hipMemcpyDeviceToHost);

        // for (uint32_t i = 0; i <= seqLen-kmerSize; i++) {
        //     printf("SA[%u]=%lu\n", i, SA_final[i]);
        // }

        
        hipMemcpy(done2, done, sizeof(size_t), hipMemcpyDeviceToHost);
        // printf("done2 = %zu, iteration = %u",done2[0], iteration);
    } while(done2[0] == 0);

    
    hipMemcpy(SA_final, suffix_array, (seqLen-kmerSize+1)*sizeof(size_t), hipMemcpyDeviceToHost);

    FILE *fp;
    fp = fopen("out_ref.txt", "w");

    for (uint32_t i = 0; i <= seqLen-kmerSize; i++) {
    	fprintf(fp, "SA[%u]=%lu\n", i, SA_final[i]);
    }


    // Wait for all computation on GPU device to finish. Needed to ensure
    // correct runtime profiling results for this function.
    hipDeviceSynchronize();
}
/**
 * Prints the fist N(=numValues) values of kmer offset and position tables to
 * help with the debugging of Assignment 2
 */
void GpuSeedTable::DeviceArrays::printValues(int numValues) {
    size_t* array2 = new size_t[numValues];
    size_t* array1 = new size_t[numValues];
    size_t* array3 = new size_t[numValues];
    size_t* intermediate_array = new size_t[numValues];
    size_t* intermediate_array2 = new size_t[numValues];
    size_t* suffix_array = new size_t[numValues];
    size_t* done = new size_t[1];
    hipError_t err;

    err = hipMemcpy(array2, d_array2, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!\n");
        exit(1);
    }

    err = hipMemcpy(array1, d_array1, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }
    err = hipMemcpy(array3, d_array3, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }

    err = hipMemcpy(intermediate_array, d_intermediate_array, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }

    err = hipMemcpy(intermediate_array2, d_intermediate_array2, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }

    err = hipMemcpy(suffix_array, d_suffix_array, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }

    err = hipMemcpy(done, d_done, sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!!!\n");
        exit(1);
    }
    // printf("done = %zu", done[0]);
    // printf("i\tkmerOffset[i]\tkmerPos2[i]\n");
    // for (int i=0; i<numValues; i++) {
    //     printf("%i\t%zu\t%zu\n", i, suffix_array[i],array2[i]);
    // }
}

